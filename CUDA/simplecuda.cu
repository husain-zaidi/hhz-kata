
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float *y)
{
    for (int i; i<n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20; // 1M elements

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // run kernel
    // add(N, x, y);
    add<<<1,1>>>(N, x, y);
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for(int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    
    std::cout << "Max error: " << maxError << std::endl;

    // free memory
    hipFree(x);
    hipFree(y);

    return 0;
}